
#include <hip/hip_runtime.h>
#include <iostream> 
#include <cstddef>
#include <chrono>
#include "hipblas.h"
#define BLOCK_SIZE 2
using namespace std;

__global__ void matmul_naive_1(double *a, double *b, double *c, size_t matrix_dim) {
  // compute position in C that this thread is responsible for
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < matrix_dim && y < matrix_dim) {
        float tmp = 0.0;
        for (int i = 0; i < matrix_dim; ++i) {
            tmp += a[x * matrix_dim + i] * b[i * matrix_dim + y];
        }
        c[x * matrix_dim + y] = tmp;
    }
}

__global__ void matmul_naive_2(double *a, double *b, double *c, size_t matrix_dim) {
  // compute position in C that this thread is responsible for
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < matrix_dim && y < matrix_dim) {
        float tmp = 0.0;
        for (int i = 0; i < matrix_dim; ++i) {
            tmp += a[y * matrix_dim + i] * b[i * matrix_dim + x];
        }
        c[y * matrix_dim + x] = tmp;
    }
}

__global__ void matmul_block(double *a, double *b, double *c, size_t matrix_dim) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x; //row
    const int y = blockIdx.y * blockDim.y + threadIdx.y; //column
    const int N_BLOCKS = matrix_dim / BLOCK_SIZE;

    __shared__ double a_temp[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double b_temp[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ double c_temp[BLOCK_SIZE*BLOCK_SIZE];
    c_temp[threadIdx.y + threadIdx.x] = 0.0;

    for (int k = 0; k != N_BLOCKS; k++){
        a_temp[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 
        a[y * matrix_dim + (threadIdx.x + BLOCK_SIZE * k)];

        b_temp[threadIdx.y * BLOCK_SIZE + threadIdx.x] = 
        b[(threadIdx.y + BLOCK_SIZE * k) * matrix_dim + x];

        __syncthreads();

        float tmp = 0.0;
        for (int i = 0; i != BLOCK_SIZE; ++i) {
            tmp += a_temp[threadIdx.y * BLOCK_SIZE + i] * b_temp[i * BLOCK_SIZE + threadIdx.x];
        }
        __syncthreads();
        c_temp[threadIdx.y * BLOCK_SIZE + threadIdx.x] += tmp;


    }
    c[y * matrix_dim + x] = c_temp[threadIdx.y * BLOCK_SIZE + threadIdx.x];

}

int check_if_equal(double *a, double *b, int N){

    for (size_t i = 0; i != N; i++){
        if (abs(a[i] - b[i]) > 1E-1){
            cout << a[i] <<" " <<  b[i] << " " << abs(a[i] - b[i]) << endl;
            return 1;
        }
    }
    
    return 0;
}

int check_if_equal_transpose(double *a, double *b, int matrix_dim){

    for (size_t i = 0; i != matrix_dim; i++){
        for (size_t j = 0; j != matrix_dim; j++){

            if (abs(a[j * matrix_dim + i] - b[i * matrix_dim + j]) > 1E-1){
                cout << a[j * matrix_dim + i] <<" " <<  b[i * matrix_dim + j] <<
                 " " << abs(a[j * matrix_dim + i] - b[i * matrix_dim + j]) << endl;
                return 1;
            }

        }
    }
    
    return 0;
}

void print_array(string array_name, double *a, int N){
    cout << array_name + " = (";
    for (size_t i = 0; i != N; i++){
        cout << a[i];
        if (i != N-1) cout << ",";
        if (i > 20) {
            cout << "...";
            break;
        }
    }
    cout << ")" << endl;
}

void print_matrix(string array_name, double *a, int N){
    //cout << array_name + " = ";
    for (size_t i = 0; i != N; i++){
        for (size_t j = 0; j != N; j++){

            cout << a[i * N + j] << " ";

        }

        cout << endl;
    }
}

void print_matrix_transpose(string array_name, double *a, int N){
    //cout << array_name + " = ";
    for (size_t i = 0; i != N; i++){
        for (size_t j = 0; j != N; j++){

            cout << a[j * N + i] << " ";

        }

        cout << endl;
    }
}

void random_fill(double *array, int N, double random_lowest, double random_highest){

    const long max_rand = 1000000L;
    static double timep = 0.0;
    timep += 1.0;
    srandom(time(NULL) + timep);
    for (size_t i = 0; i != N; ++i){
    array[i] = random_lowest+(random_highest - random_lowest)*(random() % max_rand)/max_rand;
    }
}

void matmul_using_device(double *a, double *b, double *c, size_t matrix_dim,
void (*func)(double *a, double *b, double *c, size_t matrix_dim)){
    double *d_a, *d_b, *d_c; // device copies of a, b
    size_t array_dim = pow(matrix_dim,2); 
    size_t size = array_dim * sizeof(double);

    dim3 gridDim(matrix_dim / BLOCK_SIZE, matrix_dim / BLOCK_SIZE, 1);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    // Alloc space for device copies of a, b
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    // Launch add() kernel on GPU with N blocks
    func<<<gridDim,blockDim>>>(d_a, d_b, d_c, matrix_dim);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a); hipFree(d_b); hipFree(d_c); 

}

void matmul_simple_using_host(double *a, double *b, double *c, size_t N){

    for (size_t x = 0; x != N; x++){
        for (size_t y = 0; y != N; y++){
            float tmp = 0.0;
            for (int i = 0; i != N; ++i) {
                tmp += a[x * N + i] * b[i * N + y];
            }
            c[x * N + y] = tmp;
        }
    }
}

int main() {

    double *a, *b, *c_host, *c_device; // host copies of a, b, c
    double *dev_a, *dev_b, *dev_c;
    const size_t matrix_dim = 32; // size of the arrays
    const size_t array_dim = pow(matrix_dim, 2);
    double random_lowest = 1.0; //lowest possible random double
    double random_highest = 5.0; //highest possible random double
    double alpha = 1.0;
    double beta = 0.0;
    hipblasHandle_t handle;
    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;

    size_t size = array_dim * sizeof(double);

    // Alloc space for host copies of a, b, c and setup input values
    a        = (double *)malloc(size);
    b        = (double *)malloc(size);
    c_host   = (double *)malloc(size);
    c_device = (double *)malloc(size);

    random_fill(a, array_dim, random_lowest, random_highest);
    random_fill(b, array_dim, random_lowest, random_highest);
    
    // a[0] = 1.0;
    // a[1] = 2.0;
    // a[2] = 3.0;
    // a[3] = 4.0;

    // b[0] = 9.0;
    // b[1] = 8.0;
    // b[2] = 6.0;
    // b[3] = 7.0;

    // cout << "Matrix a:" << endl;
    // print_matrix("a", a, matrix_dim);
    // cout << "Matrix b:" << endl;
    // print_matrix("b", b, matrix_dim);

    //host mm
    begin = std::chrono::steady_clock::now();
    matmul_simple_using_host(a, b, c_host, matrix_dim);
    end = std::chrono::steady_clock::now();
    std::cout << "host (sec) = " <<  (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) /1000000.0  <<std::endl;
    
    //cuda naive 1
    begin = std::chrono::steady_clock::now();
    matmul_using_device(a, b, c_device, matrix_dim, &matmul_naive_1);
    end = std::chrono::steady_clock::now();
    std::cout << "matmul_naive_1 (sec) = " <<  (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) /1000000.0  <<std::endl;
    
    if (check_if_equal(c_device, c_host, array_dim)){
        cout << "Results from host and device are not equal!" << endl;
    }
    else {
        cout << "Results from host and device are equal!" << endl;
    }


    //cuda naive 2
    begin = std::chrono::steady_clock::now();
    matmul_using_device(a, b, c_device, matrix_dim, &matmul_naive_2);
    end = std::chrono::steady_clock::now();
    std::cout << "matmul_naive_2 (sec) = " <<  (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) /1000000.0  <<std::endl;
    
    if (check_if_equal(c_device, c_host, array_dim)){
        cout << "Results from host and device are not equal!" << endl;
    }
    else {
        cout << "Results from host and device are equal!" << endl;
    }

    //cuda block
    begin = std::chrono::steady_clock::now();
    matmul_using_device(a, b, c_device, matrix_dim, &matmul_block);
    end = std::chrono::steady_clock::now();
    std::cout << "matmul_block (sec) = " <<  (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) /1000000.0  <<std::endl;
    
    if (check_if_equal(c_device, c_host, array_dim)){
        cout << "Results from host and device are not equal!" << endl;
    }
    else {
        cout << "Results from host and device are equal!" << endl;
    }


    begin = std::chrono::steady_clock::now();
    // Alloc space for device copies of a, b
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    // Copy inputs to device
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    hipblasCreate(&handle);
    hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, matrix_dim, matrix_dim, matrix_dim,
    &alpha, dev_a, matrix_dim, dev_b, matrix_dim, &beta, dev_c, matrix_dim);

    // Copy result back to host
    hipMemcpy(c_device, dev_c, size, hipMemcpyDeviceToHost);

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
    end = std::chrono::steady_clock::now();
    std::cout << "cublas (sec) = " <<  (std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count()) /1000000.0  <<std::endl;

    if (check_if_equal_transpose(c_device, c_host, matrix_dim)){
        cout << "Results from host and device are not equal!" << endl;
    }
    else {
        cout << "Results from host and device are equal!" << endl;
    }
    
    // cout << "Matrix c=a*b on host:" << endl;
    // print_matrix("c_host", c_host, matrix_dim);

    // cout << "Matrix c=a*b on device:" << endl;
    // print_matrix_transpose("c_device", c_device, matrix_dim);



    free(a); free(b); free(c_host); free(c_device);
    return 0;
}